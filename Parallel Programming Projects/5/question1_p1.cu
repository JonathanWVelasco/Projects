
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <omp.h>
#include <iostream>

// See values of N in assignment instructions.
#define N 100000
// N = { 100, 1000, 10000, 100000 }

// Do not change the seed, or your answer will not be correct
#define SEED 72

// For GPU implementation
#define BLOCKSIZE 1024

using namespace std;

struct pointData{
    double x;
    double y;
};
    
void generateDataset(struct pointData * data);
void warmUpGPU();
__global__ void getDistance(pointData *data, unsigned int *count, double *epsilon); // add parameters 


int main(int argc, char *argv[])
{
    
    
    //Read epsilon distance from command line
    if (argc!=2)
    {
        printf("\nIncorrect number of input parameters. Please input an epsilon distance.\n");
        return 0;
    }
    
    
    char inputEpsilon[20];
    strcpy(inputEpsilon,argv[1]);
    double epsilon=atof(inputEpsilon);

    //generate dataset:
    struct pointData * data;
    data=(struct pointData *)malloc(sizeof(struct pointData) * N);
    printf("\nSize of dataset (MiB): %f",(2.0*sizeof(double)*N*1.0)/(1024.0*1024.0));
    generateDataset(data);

    omp_set_num_threads(1);

    double tstart = omp_get_wtime();

    //Write your code here:
    //The data you need to use is stored in the variable "data", 
    //which is of type pointData

    warmUpGPU();
    
    // distance counts
    struct pointData *devData;
    unsigned int *totalCount;
    unsigned int *dev_totalCount;
    double *devEpsilon;
    totalCount = (unsigned int *)malloc(sizeof(unsigned int));
    // dev_totalCount = (unsigned int *)malloc(sizeof(unsigned int));
    *totalCount = 0;

    // init error
    hipError_t errCode=hipSuccess;
    if(errCode != hipSuccess){
        cout << "\nLast error: " << errCode << endl; 	
    }

    // allocate on the device: pointData
    errCode = hipMalloc((struct pointData**)&devData, sizeof(struct pointData) * N);
    if(errCode != hipSuccess) {
        cout << "\nError: point data error with code " << errCode << endl; 
    }

    // epsilon value
    errCode = hipMalloc((double**)&devEpsilon, sizeof(double));
    if(errCode != hipSuccess) {
        cout << "\nError: epsilon error with code " << errCode << endl; 
    }

    // epsilon value
    errCode = hipMalloc((unsigned int**)&dev_totalCount, sizeof(unsigned int)*N);
    if(errCode != hipSuccess) {
        cout << "\nError: count error with code " << errCode << endl; 
    }

    // copy data
    errCode = hipMemcpy( devData, data, sizeof(pointData) * N, hipMemcpyHostToDevice);
    if(errCode != hipSuccess) {
        cout << "\nError: devData memcpy error with code " << errCode << endl; 
    }

    errCode = hipMemcpy( dev_totalCount, totalCount, sizeof(unsigned int), hipMemcpyHostToDevice);
    if(errCode != hipSuccess) {
        cout << "\nError: devCounter memcpy error with code " << errCode << endl; 
    }

    errCode = hipMemcpy( devEpsilon, (void *)&epsilon, sizeof(double), hipMemcpyHostToDevice);
    if(errCode != hipSuccess) {
        cout << "\nError: devEpsilon memcpy error with code " << errCode << endl; 
    }
    
    // calculate blocks
    const unsigned int totalBlocks = ceil(N * 1.0 / 1024.0); 

    // time kernel
    double kerStart = omp_get_wtime();

    // execute kernel
    getDistance<<<totalBlocks, BLOCKSIZE>>>(devData, dev_totalCount, devEpsilon);
    hipDeviceSynchronize();

    // end kernel timer
    double kerEnd = omp_get_wtime();
    
    // copy count back out
    hipMemcpy(totalCount, dev_totalCount, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // total end timer
	double tend = omp_get_wtime();

    // Display statements
    printf("\nEpsilon Value: %f", epsilon);
    printf("\nDataset Size: %d", N);
    printf("\nTotal number of points within epsilon: %u", *totalCount);
    printf("\nTotal time (s): %f", (tend - tstart));
    printf("\nKernal time (s): %f", (kerEnd - kerStart));

	free(data);
	printf("\n");
	return 0;
}


// Do not modify the dataset generator or you will get the wrong answer
void generateDataset(struct pointData * data)
{
	// seed RNG
	srand(SEED);

	for (unsigned int i=0; i<N; i++){
		data[i].x=1000.0*((double)(rand()) / RAND_MAX);	
		data[i].y=1000.0*((double)(rand()) / RAND_MAX);	
	}
}

__global__ void getDistance(pointData *data, unsigned int *count, double *epsilon) {
    unsigned int tid = threadIdx.x + (blockIdx.x*BLOCKSIZE); 
    if(tid >= N){
        return;
    }
    for(int i = 0; i < N; i++){
        if(sqrt( (data[tid].x - data[i].x) * (data[tid].x - data[i].x) + 
          (data[tid].y - data[i].y) * (data[tid].y - data[i].y)  ) <= *epsilon){
            atomicAdd(count, int(1));
        }
    }

    return;

}


/// other warmup

__global__ void warmup(unsigned int * tmp) {
    if (threadIdx.x==0)
    *tmp=555;
    
    return;
}
    
    
    
void warmUpGPU(){
    printf("\nWarming up GPU for time trialing...\n");	
    unsigned int * dev_tmp;
    unsigned int * tmp;
    tmp=(unsigned int*)malloc(sizeof(unsigned int));
    *tmp=0;
    hipError_t errCode=hipSuccess;
    errCode=hipMalloc((unsigned int**)&dev_tmp, sizeof(unsigned int));	
    if(errCode != hipSuccess) {
        cout << "\nError: dev_tmp error with code " << errCode << endl; 
    }

    warmup<<<1,256>>>(dev_tmp);

    //copy data from device to host 
    errCode=hipMemcpy( tmp, dev_tmp, sizeof(unsigned int), hipMemcpyDeviceToHost);
    if(errCode != hipSuccess) {
        cout << "\nError: getting tmp result form GPU error with code " << errCode << endl; 
    }

        printf("\ntmp (changed to 555 on GPU): %d",*tmp);

    hipFree(dev_tmp);

    return;
}
    
